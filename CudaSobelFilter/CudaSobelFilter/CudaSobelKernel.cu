#include "hip/hip_runtime.h"
extern "C"
{
	#include "CudaSobelKernel.h"
};

const int TILE_WIDTH	= 16;
const int TILE_HEIGHT	= 16;
const int FILTER_RADIUS = 3; //  3 for averge, 1 for sobel 
const int FILTER_AREA	= (2*FILTER_RADIUS+1) * (2*FILTER_RADIUS+1);
const int BLOCK_WIDTH	= TILE_WIDTH + 2 * FILTER_RADIUS;
const int BLOCK_HEIGHT	= TILE_HEIGHT + 2 * FILTER_RADIUS;

/* DEVICE Memory */
BYTE* d_LumaPixelsIn = NULL;
BYTE* d_LumaPixelsOut = NULL;

// frame size
unsigned int* d_Width = NULL;
unsigned int* d_Height = NULL;


/* HOST Memory */
unsigned int	h_Width;
unsigned int	h_Height;
long			h_DataLength;

__global__ void SobelFilter(BYTE* g_DataIn, BYTE* g_DataOut, unsigned int width, unsigned int height);
void SobelFilterWrapper();


bool CUDAInit(unsigned int width, unsigned height)
{
	//testing !! ����Ƿ���CUDA�豸��

	h_Width = width;
	h_Height = height;

	unsigned int bufferSize = width * height * 2;

	if(d_LumaPixelsIn == NULL)
	{
		if(hipMalloc((void**)&d_LumaPixelsIn, sizeof(BYTE) * bufferSize) != hipSuccess)
			return false;
	}

	if(d_LumaPixelsOut == NULL)
	{
		if(hipMalloc((void**)&d_LumaPixelsOut, sizeof(BYTE) * bufferSize) != hipSuccess)
			return false;
	}

	if(d_Width == NULL && d_Height == NULL)
	{
		if(hipMalloc(&d_Width, sizeof(unsigned int)) != hipSuccess || hipMalloc(&d_Height, sizeof(unsigned int)) != hipSuccess)
			return false;

		hipMemcpy(d_Width,  &width, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(d_Height, &height, sizeof(unsigned int), hipMemcpyHostToDevice);
	}

	return true;
}

void CUDARelease()
{
	hipFree(d_LumaPixelsIn);
	hipFree(d_LumaPixelsOut);
	hipFree(d_Width);
	hipFree(d_Height);
}

bool CUDABeginDetection(BYTE* pImageIn, long dataLength)
{
	h_DataLength = dataLength;

	if(hipMemcpy(d_LumaPixelsIn, pImageIn, sizeof(BYTE) * dataLength, hipMemcpyHostToDevice) != hipSuccess)
		return false;

	SobelFilterWrapper();

	return true;
}

bool CUDAEndDetection(BYTE* pImageOut)
{
	if(hipMemcpy(pImageOut, d_LumaPixelsOut, sizeof(BYTE) * h_DataLength, hipMemcpyDeviceToHost) != hipSuccess)
		return false;

	return true;
}

void SobelFilterWrapper()
{
	unsigned int gridWidth = (h_Width + TILE_WIDTH - 1) / TILE_WIDTH;
	unsigned int gridHeight = (h_Height + TILE_HEIGHT - 1) / TILE_HEIGHT;
	
	dim3 dimGrid(gridWidth, gridHeight);
	dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);

	SobelFilter<<< dimGrid, dimBlock >>>(d_LumaPixelsIn, d_LumaPixelsOut, *d_Width, *d_Height);
	
	hipDeviceSynchronize();
}


__global__ void SobelFilter(BYTE* g_DataIn, BYTE* g_DataOut, unsigned int width, unsigned int height)
{
	__shared__ int sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

	int x = blockIdx.x * TILE_WIDTH + threadIdx.x - FILTER_RADIUS;
	int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - FILTER_RADIUS;

	//Clamp to the center
	x = max(FILTER_RADIUS, x);
	x = min(x, width - FILTER_RADIUS - 1);
	y = max(FILTER_RADIUS, y);
	y = min(y, height - FILTER_RADIUS - 1);

	unsigned int index = y * width + x;
	unsigned int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;

	sharedMem[sharedIndex] = g_DataIn[index];

	__syncthreads();

	if(		threadIdx.x >= FILTER_RADIUS && threadIdx.x < BLOCK_WIDTH - FILTER_RADIUS 
		&&	threadIdx.y >= FILTER_RADIUS && threadIdx.y < BLOCK_HEIGHT - FILTER_RADIUS)
	{
		float sum = 0;

		for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; ++dy)
		for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; ++dx)
		{
			float pixelValue = sharedMem[sharedIndex + (dy * blockDim.x + dx)];
			sum += pixelValue;
		}

		g_DataOut[index] = sum / FILTER_AREA;
	}
}
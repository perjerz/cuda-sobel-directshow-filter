#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// File: CudaSobelKernel.cu
// 
// Author: Ren Yifei
// 
// Desc:
//
//
//------------------------------------------------------------------------------

extern "C"
{
	#include "CudaSobelKernel.h"
	#include <stdio.h>
};

//////////////////////////////////////////////////////////////////////////
FILE* fout;
#define MYLOG(x) {fout=fopen("c:\\dbg.txt","a");fprintf(fout,"value: %d\n\n",x);fclose(fout);}
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

//BYTE* h_LogData = -1;

//#define LogCuda(x) {CUDA_SAFE_CALL( hipMemcpy(h_LogData, x, sizeof(x), hipMemcpyDeviceToHost) );\
	//MYLOG(h_LogData)\
}

typedef int TEST_TYPE;

TEST_TYPE* d_kerneltest;


//////////////////////////////////////////////////////////////////////////


const int TILE_WIDTH	= 16;
const int TILE_HEIGHT	= 16;
const int FILTER_RADIUS = 3; //  3 for averge, 1 for sobel 
const int FILTER_AREA	= (2*FILTER_RADIUS+1) * (2*FILTER_RADIUS+1);
const int BLOCK_WIDTH	= TILE_WIDTH + 2 * FILTER_RADIUS;
const int BLOCK_HEIGHT	= TILE_HEIGHT + 2 * FILTER_RADIUS;

// CUDA Context

//hipCtx_t* sobelCtx;

/* DEVICE Memory */
BYTE* d_LumaPixelsIn = NULL;
BYTE* d_LumaPixelsOut = NULL;

// frame size
int* d_Width = NULL;
int* d_Height = NULL;


/* HOST Memory */
int	h_Width;
int	h_Height;
long h_DataLength;

__global__ void SobelFilter(BYTE* g_DataIn, BYTE* g_DataOut, int* width, int* height, TEST_TYPE* d_kt);

void SobelFilterWrapper(BYTE* pImageIn);


bool CUDAInit(int width, int height)
{
	//testing !! ����Ƿ���CUDA�豸!

	//if(hipInit(0)!=hipSuccess)
		//return false;

	//hipDeviceGet(&cuDevice, 0);
	//hipCtxCreate(&cuCtx, 0, cuDevice);


	h_Width = width;
	h_Height = height;

// 	MYLOG(h_Width);
// 	MYLOG(h_Height);

	return true;
}

void CUDARelease()
{
	CUDA_SAFE_CALL( hipFree(d_LumaPixelsIn) );
	CUDA_SAFE_CALL( hipFree(d_LumaPixelsOut) );
	CUDA_SAFE_CALL( hipFree(d_Width) );
	CUDA_SAFE_CALL( hipFree(d_Height) );
}

bool CUDABeginDetection(BYTE* pImageIn, long dataLength)
{
	//////////////////////////////////////////////////////////////////////////
	CUDA_SAFE_CALL( hipMalloc(&d_kerneltest, sizeof(TEST_TYPE)) );
	//////////////////////////////////////////////////////////////////////////


	h_DataLength = dataLength;


	if(d_Width == NULL && d_Height == NULL)
	{
		CUDA_SAFE_CALL( hipMalloc(&d_Width, sizeof(int)) );
		CUDA_SAFE_CALL( hipMalloc(&d_Height, sizeof(int)) );

		CUDA_SAFE_CALL( hipMemcpy(d_Width,  &h_Width, sizeof(int), hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL( hipMemcpy(d_Height, &h_Height, sizeof(int), hipMemcpyHostToDevice) );
	}

	if(d_LumaPixelsIn == NULL)
	{
		CUDA_SAFE_CALL( hipMalloc((void**)&d_LumaPixelsIn, sizeof(BYTE) * h_DataLength / 2) );
	}

	if(d_LumaPixelsOut == NULL)
	{
		CUDA_SAFE_CALL( hipMalloc((void**)&d_LumaPixelsOut, sizeof(BYTE) * h_DataLength / 2) );
	}

	CUDA_SAFE_CALL( hipMemcpy((void*)d_LumaPixelsIn, (void*)pImageIn, sizeof(BYTE) * h_DataLength / 2, hipMemcpyHostToDevice) );

	SobelFilterWrapper(pImageIn);

	return true;
}

bool CUDAEndDetection(BYTE* pImageOut)
{
	CUDA_SAFE_CALL( hipMemcpy(pImageOut, d_LumaPixelsOut, sizeof(BYTE) * h_DataLength / 2, hipMemcpyDeviceToHost) );

	return true;
}

void SobelFilterWrapper(BYTE* pImageIn)
{
	int gridWidth = (h_Width + TILE_WIDTH - 1) / TILE_WIDTH;
	int gridHeight = (h_Height + TILE_HEIGHT - 1) / TILE_HEIGHT;
	
	dim3 dimGrid(gridWidth, gridHeight);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

	SobelFilter<<< dimGrid, dimBlock >>>(d_LumaPixelsIn, d_LumaPixelsOut, d_Width, d_Height, d_kerneltest);	

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}


__global__ void SobelFilter(BYTE* g_DataIn, BYTE* g_DataOut, int* width, int* height, TEST_TYPE* d_kt)
{
	__shared__ BYTE sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

	int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
	int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

	//No filtering for the edges
	x = max(FILTER_RADIUS, x);
	x = min(x, *width  - FILTER_RADIUS - 1);
	y = max(FILTER_RADIUS, y);
	y = min(y, *height - FILTER_RADIUS - 1);

	int index = y * (*width) + x;
	int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;

	sharedMem[sharedIndex] = g_DataIn[index];

	__syncthreads();

	if(		threadIdx.x >= FILTER_RADIUS && threadIdx.x < BLOCK_WIDTH - FILTER_RADIUS 
		&&	threadIdx.y >= FILTER_RADIUS && threadIdx.y < BLOCK_HEIGHT - FILTER_RADIUS)
	{
		float sum = 0;

		for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; ++dy)
		for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; ++dx)
		{
			float pixelValue = (float)(sharedMem[sharedIndex + (dy * blockDim.x + dx)]);

			sum += pixelValue;
		}

		BYTE res = (BYTE)(sum / FILTER_AREA);

		g_DataOut[index] = res;
	}	
}